/*
 how to run nvcc cuda_threads.cu -o cuda_threads
 ./cuda_threads
*/
#include "hip/hip_runtime.h"


#include <cstdio>

// Kerner which will be ran on the GPU also know as device Code
__global__ void HelloGPU()
{
	printf("Hello World from the GPU\n");
}

// in the main we have host code, code that would be ran on CPU also known as Host code
int main()
{
    /*calling the kernel with execution cofiguration 
     <<< NUMBER_OF_BLOCKS, NUMBER_OF_THREADS_PER_BLOCK>>>
     <<<1,1>>> means 1 block which contains 1 thread.
        the code would be executed once, in a block using a thread
     <<<1,5>>> means 1 block which contains 5 threads, 
        the code would be executed 5 times by 5 different threads
    */
	HelloGPU<<<1,5>>>();
	hipDeviceSynchronize();
	
	getchar();
	return 0;
}